
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void add (const int *a, const int *b, int *c)
{
    *c = *a + *b; // a, b, c must be on the GPU memory -> we need to allocate memory on the GPU
}

int main()
{
    int a, b, c;
    int *d_a, *d_b, *d_c;
    int size = sizeof(int);
    // Allocate space in Device memory
    hipMalloc(&d_a, size);
    hipMalloc(&d_b, size);
    hipMalloc(&d_c, size);
    // Input values
    a = 2;
    b = 7;
    // Copy inputs to Device
    hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice); // destination, origin, how much memory
    hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);
    hipMemcpy(d_c, &c, size, hipMemcpyHostToDevice);
    // Launch add() on the Device
    add<<<1,1>>>(d_a, d_b, d_c);
    // Result is in d_c, we need to copy it back
    hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);
    // Cleanup
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    return 0;
}